#include "hip/hip_runtime.h"
/************************************************************
********
* MatrixMulti CUDA program.
************************************************************
*********/
#define BLOCK_SIZE 16
#define WIDTH (BLOCK_SIZE * 128)
#define HEIGHT (BLOCK_SIZE * 128)

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_cuda_drvapi.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_cuda_gl.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/hip/hip_runtime_api.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_functions.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_image.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/hip/hip_vector_types.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_string.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_timer.h"

typedef struct {
int width;
int height;
float* elements;
} Matrix;
/************************************************************
************/
/* Init CUDA                                                                                                                    */
/************************************************************
************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
        int count = 0;
        int i = 0;

        hipGetDeviceCount(&count);
        if(count == 0) {
                fprintf(stderr, "There is no device.\n");
                return false;
        }

        for(i = 0; i < count; i++) {
                hipDeviceProp_t prop;
                if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
                        if(prop.major >= 1) {
                                break;
                        }
                }
        }
        if(i == count) {
                fprintf(stderr, "There is no device supporting CUDA.\n");
                return false;
        }
        hipSetDevice(i);

        printf("CUDA initialized.\n");
        return true;
}

#endif



// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
        for (int i = 0; i < size; ++i)
                data[i] = rand() / (float)RAND_MAX;

        
}

/************************************************************
************/
//Kernel

//Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int rep=0; rep<16; rep++)
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e]
			* B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}


/************************************************************
************/
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load A and B to device memory
	Matrix d_A,d_B,d_C;
	struct timeval tv1, tv2;

	size_t size = A.width * A.height * sizeof(float);

	d_A.width =A.width; d_A.height = A.width;
	d_B.width = B.width; d_B.height = B.height;
	d_C.width = C.width; d_C.height = C.height;

	hipMalloc((void**)&d_A.elements, size);
	hipMalloc((void**)&d_B.elements, size);
	hipMalloc((void**)&d_C.elements, size);
	gettimeofday(&tv1, NULL);
	hipMemcpy(d_A.elements,A.elements, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	gettimeofday(&tv2, NULL);
        printf("copying takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));

	// Allocate C in device memory

	size = C.width * C.height * sizeof(float);


	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	gettimeofday(&tv1, NULL);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	double x;
	for (int i=0;i<size*48;i++)
		x = x*x - i;


	hipDeviceSynchronize();
	gettimeofday(&tv2, NULL);
	printf("Invoking kernel takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));

	gettimeofday(&tv1, NULL);
	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size,hipMemcpyDeviceToHost);
	gettimeofday(&tv2, NULL);
	printf("Copying C takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));

	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}


/************************************************************
************/
/*MAIN                                                                                                            */
/************************************************************
************/
int main(int argc, char* argv[])
{
	if(!InitCUDA()) {
		    return 0;
	}

	// allocate host memory for matrices A and B
	Matrix h_A,h_B,h_C;
	h_A.width=WIDTH;
	h_A.height=HEIGHT;
	h_B.width=WIDTH;
	h_B.height=HEIGHT;
	h_C.width=WIDTH;
	h_C.height=HEIGHT;

	unsigned int size = WIDTH*HEIGHT;
	unsigned int mem_size = sizeof(float) * size;

	//h_A.elements= (float*) malloc(mem_size);
	//h_B.elements= (float*) malloc(mem_size);
	//h_C.elements= (float*) malloc(mem_size);
	hipHostAlloc(&h_A.elements, mem_size, hipHostMallocDefault);
	hipHostAlloc(&h_B.elements, mem_size, hipHostMallocDefault);
	hipHostAlloc(&h_C.elements, mem_size, hipHostMallocDefault);

	// set seed for rand()
	srand(2006);

	// initialize host memory
	randomInit(h_A.elements, size);
	randomInit(h_B.elements, size);

	//invoke MatMul
	MatMul(h_A,h_B,h_C);

	return 0;
}
