#include "hip/hip_runtime.h"
/************************************************************
********
* MatrixMulti CUDA program.
************************************************************
*********/
#define BLOCK_SIZE 4
#define WIDTH (BLOCK_SIZE * 128)
#define HEIGHT (BLOCK_SIZE * 128)

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
//#include <cutil.h>
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_cuda_drvapi.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_cuda_gl.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/hip/hip_runtime_api.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_functions.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_image.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/hip/hip_vector_types.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_string.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_timer.h"

typedef struct {
int width;
int height;
float* elements;
} Matrix;
/************************************************************
************/
/* Init CUDA                                                                                                                    */
/************************************************************
************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
        int count = 0;
        int i = 0;

        hipGetDeviceCount(&count);
        if(count == 0) {
                fprintf(stderr, "There is no device.\n");
                return false;
        }

        for(i = 0; i < count; i++) {
                hipDeviceProp_t prop;
                if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
                        if(prop.major >= 1) {
                                break;
                        }
                }
        }
        if(i == count) {
                fprintf(stderr, "There is no device supporting CUDA.\n");
                return false;
        }
        hipSetDevice(i);

        printf("CUDA initialized.\n");
        return true;
}

#endif



// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
        for (int i = 0; i < size; ++i)
                data[i] = rand() / (float)RAND_MAX;

        
}

/************************************************************
************/
//Kernel

//Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int rep=0; rep<512; rep++)
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e]
			* B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}


/************************************************************
************/
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load A and B to device memory
	Matrix d_A,d_B,d_C;
	struct timeval tv1, tv2;
	static hipStream_t stream = NULL;

	size_t size = A.width * A.height * sizeof(float);

	d_A.width =A.width; d_A.height = A.width;
	d_B.width = B.width; d_B.height = B.height;
	d_C.width = C.width; d_C.height = C.height;

	hipStreamCreate(&stream);
	hipMalloc((void**)&d_A.elements, size);
	hipMalloc((void**)&d_B.elements, size);
	hipMalloc((void**)&d_C.elements, size);

	gettimeofday(&tv1, NULL);
	hipMemcpyAsync(d_A.elements,A.elements, size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_B.elements, B.elements, size, hipMemcpyHostToDevice, stream);
	//hipMemcpy(d_A.elements,A.elements, size, hipMemcpyHostToDevice);
	//hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	gettimeofday(&tv2, NULL);
        printf("P1 copying takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));
	sleep(1);

	// Allocate C in device memory

	size = C.width * C.height * sizeof(float);


	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	
	hipConfigureCall(dimGrid, dimBlock, 0, stream);
	gettimeofday(&tv1, NULL);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	hipDeviceSynchronize();
	gettimeofday(&tv2, NULL);
	printf("P1 Invoking kernel takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));

	gettimeofday(&tv1, NULL);
	// Read C from device memory
	hipMemcpyAsync(C.elements, d_C.elements, size,hipMemcpyDeviceToHost, stream);
	gettimeofday(&tv2, NULL);
	printf("P1 copying C takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));

	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipStreamDestroy(stream);
}


/************************************************************
************/
/*MAIN                                                                                                            */
/************************************************************
************/
int main(int argc, char* argv[])
{
	if(!InitCUDA()) {
		    return 0;
	}

	// allocate host memory for matrices A and B
	Matrix h_A,h_B,h_C;
	h_A.width=WIDTH;
	h_A.height=HEIGHT;
	h_B.width=WIDTH;
	h_B.height=HEIGHT;
	h_C.width=WIDTH;
	h_C.height=HEIGHT;

	unsigned int size = WIDTH*HEIGHT;
	unsigned int mem_size = sizeof(float) * size;

	//h_A.elements= (float*) malloc(mem_size);
	//h_B.elements= (float*) malloc(mem_size);
	//h_C.elements= (float*) malloc(mem_size);
	hipHostAlloc(&h_A.elements, mem_size, hipHostMallocDefault);
	hipHostAlloc(&h_B.elements, mem_size, hipHostMallocDefault);
	hipHostAlloc(&h_C.elements, mem_size, hipHostMallocDefault);

	// set seed for rand()
	srand(2006);

	// initialize host memory
	randomInit(h_A.elements, size);
	randomInit(h_B.elements, size);

	//invoke MatMul
	MatMul(h_A,h_B,h_C);

	return 0;
}
