#include "hip/hip_runtime.h"
/************************************************************
********
* MatrixMulti CUDA program.
************************************************************
*********/
#define BLOCK_SIZE 4
#define WIDTH (BLOCK_SIZE * 128)
#define HEIGHT (BLOCK_SIZE * 128)

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
//#include <cutil.h>
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_cuda_drvapi.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_cuda_gl.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/hip/hip_runtime_api.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_functions.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_image.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/hip/hip_vector_types.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_string.h"
#include "/home/syma/etc/CUDA_5.0_SAMPLES/common/inc/helper_timer.h"

typedef struct {
int width;
int height;
float* elements;
} Matrix;
/************************************************************
************/
/* Init CUDA                                                                                                                    */
/************************************************************
************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
        int count = 0;
        int i = 0;

        hipGetDeviceCount(&count);
        if(count == 0) {
                fprintf(stderr, "There is no device.\n");
                return false;
        }

        for(i = 0; i < count; i++) {
                hipDeviceProp_t prop;
                if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
                        if(prop.major >= 1) {
                                break;
                        }
                }
        }
        if(i == count) {
                fprintf(stderr, "There is no device supporting CUDA.\n");
                return false;
        }
        hipSetDevice(i);

        printf("CUDA initialized.\n");
        return true;
}

#endif



// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
        for (int i = 0; i < size; ++i)
                data[i] = rand() / (float)RAND_MAX;

        
}

/************************************************************
************/
//Kernel

//Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int rep=0; rep<16; rep++)
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e]
			* B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

static struct timeval tv0, tv1, tv2;
/************************************************************
************/
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load A and B to device memory
	Matrix d_A,d_B,d_C;
	static hipStream_t stream = NULL;

	size_t size = A.width * A.height * sizeof(float);

	d_A.width =A.width; d_A.height = A.width;
	d_B.width = B.width; d_B.height = B.height;
	d_C.width = C.width; d_C.height = C.height;
	

	hipStreamCreate(&stream);
	gettimeofday(&tv1, NULL);
	hipMalloc((void**)&d_A.elements, size);
	hipMalloc((void**)&d_B.elements, size);
	hipMalloc((void**)&d_C.elements, size);
	gettimeofday(&tv2, NULL);
	printf("P2 hipMalloc takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));
	sleep(2);

	gettimeofday(&tv1, NULL);
	//hipMemcpyAsync(d_A.elements,A.elements, size, hipMemcpyHostToDevice, stream);
	//hipMemcpyAsync(d_B.elements, B.elements, size, hipMemcpyHostToDevice, stream);
	hipMemcpy(d_A.elements,A.elements, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	// Allocate C in device memory

	size = C.width * C.height * sizeof(float);


	// Invoke kernel
	//dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	//MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	//double x;
	//for (int i=0;i<size*48;i++)
	//	x = x*x - i;


	hipDeviceSynchronize();
	hipMemcpy(C.elements, d_C.elements, size,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	gettimeofday(&tv2, NULL);
	printf("P2 copying takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));

	// Free device memory
	gettimeofday(&tv1, NULL);
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	hipStreamDestroy(stream);
	gettimeofday(&tv2, NULL);
	printf("P2 free takes %ld micro seconds\n", (tv2.tv_sec - tv1.tv_sec) * 1000000L + (tv2.tv_usec - tv1.tv_usec));
}


/************************************************************
************/
/*MAIN                                                                                                            */
/************************************************************
************/
int main(int argc, char* argv[])
{
	if(!InitCUDA()) {
		    return 0;
	}

	gettimeofday(&tv0, NULL);
	// allocate host memory for matrices A and B
	Matrix h_A,h_B,h_C;
	h_A.width=WIDTH;
	h_A.height=HEIGHT;
	h_B.width=WIDTH;
	h_B.height=HEIGHT;
	h_C.width=WIDTH;
	h_C.height=HEIGHT;

	unsigned int size = WIDTH*HEIGHT;
	unsigned int mem_size = sizeof(float) * size;

	//h_A.elements= (float*) malloc(mem_size);
	//h_B.elements= (float*) malloc(mem_size);
	//h_C.elements= (float*) malloc(mem_size);
	hipHostAlloc(&h_A.elements, mem_size, hipHostMallocDefault);
	hipHostAlloc(&h_B.elements, mem_size, hipHostMallocDefault);
	hipHostAlloc(&h_C.elements, mem_size, hipHostMallocDefault);

	// set seed for rand()
	srand(2006);

	// initialize host memory
	randomInit(h_A.elements, size);
	randomInit(h_B.elements, size);

	//invoke MatMul
	MatMul(h_A,h_B,h_C);
	gettimeofday(&tv2, NULL);
	printf("P2 takes %ld micro seconds\n", (tv2.tv_sec - tv0.tv_sec) * 1000000L + (tv2.tv_usec - tv0.tv_usec));

	return 0;
}
