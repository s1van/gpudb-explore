#include "hip/hip_runtime.h"
// local evictions
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "gmm.h"

__global__ void kernel_inc(int *data, int count)
{
	int tot_threads = gridDim.x * blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (; i < count; i += tot_threads)
		data[i]++;
}

int test_evict_local()
{
	int *dptr = NULL, *dptr2 = NULL, *ptr = NULL;
	int count = 1000 * 1000 * 200;
	size_t size = sizeof(int) * count;
	int i, ret = 0;

	ptr = (int *)malloc(size);
	if (!ptr) {
		GMM_TPRINT("malloc failed for ptr\n");
		return -1;
	}
	memset(ptr, 0, size);

	if (hipMalloc(&dptr, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		free(ptr);
		return -1;
	}

	if (hipMalloc(&dptr2, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		hipFree(dptr);
		free(ptr);
		return -1;
	}

	if (hipMemcpy(dptr, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice to dptr failed\n");
		ret = -1;
		goto finish;
	}
	if (hipMemcpy(dptr2, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice to deptr2 failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("all hipMemcpyHostToDevice succeeded\n");

	if (cudaReference(0, HINT_DEFAULT) != hipSuccess) {
		GMM_TPRINT("cudaReference failed\n");
		ret = -1;
		goto finish;
	}
	kernel_inc<<<256, 128>>>(dptr, count);

	if (hipDeviceSynchronize() != hipSuccess) {
		GMM_TPRINT("hipDeviceSynchronize returned error\n");
		ret = -1;
		goto finish;
	}
	else
		GMM_TPRINT("kernel finished\n");

	if (cudaReference(0, HINT_DEFAULT) != hipSuccess) {
		GMM_TPRINT("cudaReference failed\n");
		ret = -1;
		goto finish;
	}
	kernel_inc<<<256, 128>>>(dptr2, count);

	if (hipDeviceSynchronize() != hipSuccess) {
		GMM_TPRINT("hipDeviceSynchronize returned error\n");
		ret = -1;
		goto finish;
	}
	else
		GMM_TPRINT("kernel finished\n");

	if (hipMemcpy(ptr, dptr, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < count; i++)
		if (ptr[i] != 1) {
			GMM_TPRINT("verification failed at ptr[%d]==%d\n", i, ptr[i]);
			ret = -1;
			goto finish;
		}

	if (hipMemcpy(ptr, dptr2, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < count; i++)
		if (ptr[i] != 1) {
			GMM_TPRINT("verification failed at ptr[%d]==%d\n", i, ptr[i]);
			ret = -1;
			goto finish;
		}

	GMM_TPRINT("verification passed\n");

finish:
	if (hipFree(dptr2) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	if (hipFree(dptr) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	free(ptr);

	return ret;
}
