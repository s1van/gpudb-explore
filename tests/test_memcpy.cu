// memory copy
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "gmm.h"

int test_memcpy()
{
	void *dptr, *dptr2, *ptr, *ptr2;
	size_t size = 1024 * 1024 * 10, i;
	int ret = 0;

	// Mallocs
	ptr = malloc(size);
	if (!ptr) {
		GMM_TPRINT("malloc failed for ptr\n");
		return -1;
	}

	ptr2 = malloc(size);
	if (!ptr2) {
		GMM_TPRINT("malloc failed for ptr2\n");
		free(ptr);
		return -1;
	}

	if (hipMalloc(&dptr, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc for dptr failed\n");
		free(ptr2);
		free(ptr);
		return -1;
	}

	if (hipMalloc(&dptr2, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc for dptr2 failed\n");
		hipFree(dptr);
		free(ptr2);
		free(ptr);
		return -1;
	}

	for(i = 0; i < size; i += 4096) {
		*((char *)ptr + i) = 'x';
		*((char *)ptr2 + i) = 'y';
	}

	if (hipMemcpy(dptr, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpy HtoD failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyHostToDevice succeeded\n");

	if (hipMemcpy(dptr2, dptr, size, hipMemcpyDeviceToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoD failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyDeviceToDevice succeeded\n");

	if (hipMemcpy(ptr2, dptr2, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < size; i += 4096)
		if (*((char *)ptr2 + i) != 'x') {
			GMM_TPRINT("verification failed at i = %lu (*ptr2 = %c)\n", i, \
					*((char *)ptr2 + i));
			ret = -1;
			goto finish;
		}

	GMM_TPRINT("verification passed\n");

finish:
	if (hipFree(dptr) != hipSuccess) {
		GMM_TPRINT("hipFree for dptr failed\n");
	}
	if (hipFree(dptr2) != hipSuccess) {
		GMM_TPRINT("hipFree for dptr2 failed\n");
	}
	free(ptr);
	free(ptr2);

	return ret;
}
