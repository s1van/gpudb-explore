#include "hip/hip_runtime.h"
// remote evictions
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "gmm.h"

__global__ void kernel_inc(int *data, int count)
{
	int tot_threads = gridDim.x * blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (; i < count; i += tot_threads)
		data[i]++;
}

int test_evict_remote()
{
	int *dptr = NULL, *ptr = NULL;
	size_t size, sfree, total;
	int i, count, ret = 0;
	//int c;

	if (hipMemGetInfo(&sfree, &total) != hipSuccess) {
		GMM_TPRINT("failed to get mem info\n");
		return -1;
	}
	size = total * 3 / 4;
	count = size / sizeof(int);

	ptr = (int *)malloc(size);
	if (!ptr) {
		GMM_TPRINT("malloc failed for ptr\n");
		return -1;
	}
	memset(ptr, 0, size);

	if (hipMalloc(&dptr, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		free(ptr);
		return -1;
	}

	if (hipMemcpy(dptr, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice to dptr failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyHostToDevice succeeded\n");

	// First kernel launch
	if (cudaReference(0, HINT_DEFAULT) != hipSuccess) {
		GMM_TPRINT("cudaReference failed\n");
		ret = -1;
		goto finish;
	}

	kernel_inc<<<256, 128>>>(dptr, count);

	if (hipDeviceSynchronize() != hipSuccess) {
		GMM_TPRINT("hipDeviceSynchronize returned error\n");
		ret = -1;
		goto finish;
	}
	else
		GMM_TPRINT("1st kernel finished\n");

	// Wait for the object being evicted by its co-runner
	//c = getchar();

	// Second kernel launch
	if (cudaReference(0, HINT_DEFAULT) != hipSuccess) {
		GMM_TPRINT("cudaReference failed\n");
		ret = -1;
		goto finish;
	}

	kernel_inc<<<256, 128>>>(dptr, count);

	if (hipDeviceSynchronize() != hipSuccess) {
		GMM_TPRINT("hipDeviceSynchronize returned error\n");
		ret = -1;
		goto finish;
	}
	else
		GMM_TPRINT("2nd kernel finished\n");

	// Copy back and do verification
	if (hipMemcpy(ptr, dptr, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < count; i++)
		if (ptr[i] != 2) {
			GMM_TPRINT("verification failed at ptr[%d]==%d\n", i, ptr[i]);
			ret = -1;
			goto finish;
		}
	GMM_TPRINT("verification passed\n");

finish:
	if (hipFree(dptr) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	free(ptr);

	return ret;
}
