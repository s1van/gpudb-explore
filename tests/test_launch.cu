#include "hip/hip_runtime.h"
// kernel launch
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "gmm.h"

__global__ void kernel_inc(int *data, int count)
{
	int tot_threads = gridDim.x * blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (; i < count; i += tot_threads)
		data[i]++;
}

int test_launch()
{
	int *dptr = NULL, *ptr = NULL, *ptr2 = NULL;
	int count = 1000 * 1000 * 200;
	size_t size = sizeof(int) * count;
	int i, ret = 0;

	ptr = (int *)malloc(size);
	if (!ptr) {
		GMM_TPRINT("malloc failed for ptr\n");
		return -1;
	}
	ptr2 = (int *)malloc(size);
	if (!ptr2) {
		GMM_TPRINT("malloc failed for ptr\n");
		free(ptr);
		return -1;
	}
	memset(ptr, 0, size);

	if (hipMalloc(&dptr, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		free(ptr);
		return -1;
	}

	if (hipMemcpy(dptr, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyHostToDevice succeeded\n");

	if (cudaReference(0, HINT_DEFAULT) != hipSuccess) {
		GMM_TPRINT("cudaReference failed\n");
		ret = -1;
		goto finish;
	}

	kernel_inc<<<256, 128>>>(dptr, count);

	if (hipDeviceSynchronize() != hipSuccess) {
		GMM_TPRINT("hipDeviceSynchronize returned error\n");
		ret = -1;
		goto finish;
	}
	else
		GMM_TPRINT("kernel finished\n");

	if (hipMemcpy(ptr2, dptr, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < count; i++)
		if (ptr2[i] != 1) {
			GMM_TPRINT("verification failed at ptr[%d]==%d\n", i, ptr[i]);
			ret = -1;
			goto finish;
		}
	GMM_TPRINT("verification passed\n");

finish:
	if (hipFree(dptr) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	free(ptr);
	free(ptr2);

	return ret;
}
