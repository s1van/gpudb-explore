// memory region allocation
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "gmm.h"

int test_malloc()
{
	void *dptr = NULL;
	size_t size = 1024;

	while (size < 1024L * 1024L * 513) {
		GMM_TPRINT("allocating %lu bytes\n", size);
		if (hipMalloc(&dptr, size) != hipSuccess) {
			GMM_TPRINT("hipMalloc failed\n");
			return -1;
		}
		GMM_TPRINT("dptr = %p\n", dptr);
		if (hipFree(dptr) != hipSuccess) {
			GMM_TPRINT("hipFree failed\n");
			return -1;
		}
		GMM_TPRINT("region freed\n");
		size *= 2;
	}

	return 0;
}
