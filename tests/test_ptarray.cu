#include "hip/hip_runtime.h"
// dptr array regions
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "gmm.h"

__global__ void kernel_inc(int **data, int dim, int count)
{
	int tot_threads = gridDim.x * blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (; i < count; i += tot_threads) {
		for (int j = 0; j < dim; j++)
			data[j][i]++;
	}
}

int test_ptarray()
{
	int **dptr = NULL, *dptr1 = NULL, *dptr2 = NULL, *ptr = NULL;
	int count = 1000 * 1000 * 10;
	size_t size = sizeof(int) * count;
	int i, ret = 0;

	ptr = (int *)malloc(size);
	if (!ptr) {
		GMM_TPRINT("malloc failed for ptr\n");
		return -1;
	}
	memset(ptr, 0, size);

	if (hipMalloc(&dptr1, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		free(ptr);
		return -1;
	}
	if (hipMalloc(&dptr2, size) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		hipFree(dptr1);
		free(ptr);
		return -1;
	}
	if (cudaMallocEx((void **)&dptr, sizeof(int *) * 2, HINT_PTARRAY) != hipSuccess) {
		GMM_TPRINT("hipMalloc failed\n");
		hipFree(dptr2);
		hipFree(dptr1);
		free(ptr);
		return -1;
	}

	if (hipMemcpy(dptr1, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice failed\n");
		ret = -1;
		goto finish;
	}
	if (hipMemcpy(dptr2, ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice failed\n");
		ret = -1;
		goto finish;
	}
	if (hipMemcpy(dptr, &dptr1, sizeof(int *), hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice failed\n");
		ret = -1;
		goto finish;
	}
	if (hipMemcpy(dptr + 1, &dptr2, sizeof(int *), hipMemcpyHostToDevice) != hipSuccess) {
		GMM_TPRINT("hipMemcpyHostToDevice failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("hipMemcpyHostToDevice succeeded\n");

	i = 1;
	do {
		if (cudaReference(0, HINT_READ | HINT_PTARRAY | HINT_PTADEFAULT) != hipSuccess) {
			GMM_TPRINT("cudaReference failed\n");
			ret = -1;
			goto finish;
		}
		kernel_inc<<<256, 128>>>(dptr, 2, count);
	} while (i-- > 0);

	if (hipDeviceSynchronize() != hipSuccess) {
		GMM_TPRINT("hipDeviceSynchronize returned error\n");
		ret = -1;
		goto finish;
	}
	else
		GMM_TPRINT("kernel finished\n");

	if (hipMemcpy(ptr, dptr1, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("first hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < count; i++)
		if (ptr[i] != 2) {
			GMM_TPRINT("verification failed at ptr[%d]==%d\n", i, ptr[i]);
			ret = -1;
			goto finish;
		}

	if (hipMemcpy(ptr, dptr2, size, hipMemcpyDeviceToHost) != hipSuccess) {
		GMM_TPRINT("hipMemcpy DtoH failed\n");
		ret = -1;
		goto finish;
	}
	GMM_TPRINT("second hipMemcpyDeviceToHost succeeded\n");

	for(i = 0; i < count; i++)
		if (ptr[i] != 2) {
			GMM_TPRINT("verification failed at ptr[%d]==%d\n", i, ptr[i]);
			ret = -1;
			goto finish;
		}

	GMM_TPRINT("verification passed\n");

finish:
	if (hipFree(dptr) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	if (hipFree(dptr2) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	if (hipFree(dptr1) != hipSuccess) {
		GMM_TPRINT("hipFree failed\n");
	}
	free(ptr);

	return ret;
}
